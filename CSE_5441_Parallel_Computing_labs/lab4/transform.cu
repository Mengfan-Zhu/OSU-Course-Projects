#include "hip/hip_runtime.h"
#include "transform.h"
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
// extern "C" void doTransform(struct work_entry *ori_queue, struct work_entry *result_queue, int count);
// call corresponding transform functions
__device__ uint16_t transCall(char cmd, uint16_t key){
    uint16_t result;
    if(cmd == 'A'){
        result = transformA(key); 
    }
    else if(cmd == 'B'){
        result = transformB(key); 
    }
    else if(cmd == 'C'){
        result = transformC(key); 
    }
    else if(cmd == 'D'){
        result = transformD(key); 
    }
    return result;
}
// kernel - producer
__global__ void transProducer(struct work_entry *work_queue, int count){
    int tid;
    tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < count){
        work_queue[tid].key = transCall(work_queue[tid].cmd, work_queue[tid].key);   
    }
}
// kernel - consumer
__global__ void transConsumer(struct work_entry *work_queue, int count){
    int tid;
    tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < count){
        work_queue[tid].key = transCall(work_queue[tid].cmd, work_queue[tid].key);   
    }
}

extern "C" void doTransform(struct work_entry *ori_queue, struct work_entry *result_queue, int count){
    // pointers for host and device memory
    struct work_entry *d_queue;
    // thread hierarchy
    int num_threads = NUM_THREAD;
    int num_blocks = count/num_threads + 1;
    time_t p_time, c_time, p_begin, p_end, c_begin, c_end;
    // allocate memory
    size_t memSize;
    memSize = count * sizeof(struct work_entry);
    hipMalloc((void**)&d_queue, memSize);
    time(&p_begin);
    // copy data
    hipMemcpy(d_queue, ori_queue, memSize, hipMemcpyHostToDevice);
    // launch kernel
    dim3 dimGrid(num_blocks);
    dim3 dimBlock(num_threads);
    // do transformation and copy results
    transProducer<<<dimGrid, dimBlock>>>(d_queue, count);
    hipMemcpy(ori_queue, d_queue, memSize, hipMemcpyDeviceToHost);
    time(&p_end);
    p_time = p_end - p_begin;
    time(&c_begin);
    transConsumer<<<dimGrid, dimBlock>>>(d_queue, count);
    hipMemcpy(result_queue, d_queue, memSize, hipMemcpyDeviceToHost);
    time(&c_end);
    c_time = c_end - c_begin;
    fprintf(stderr, "total producer time: %d\ntotal consumer time: %d\n", p_time, c_time);
    hipFree(d_queue);
}